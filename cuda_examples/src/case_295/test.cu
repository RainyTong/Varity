
/* This is a automatically generated test. Do not modify */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9) {
if (comp > (-0.0f - var_1 - var_2 - var_3)) {
  float tmp_1 = -1.4288E-21f;
comp = tmp_1 + (-1.7077E-43f * +0.0f + (var_4 * floorf((var_5 - atan2f(-1.6666E-42f, (+1.8771E-36f / var_6 - (+1.6908E-8f - (var_7 / var_8 * var_9))))))));
}
   printf("%.17g\n", comp);

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  return ret;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10);
  hipDeviceSynchronize();

  return 0;
}